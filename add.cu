
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{

	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}


typedef struct {
	float x, y, z;
} Vec3;

typedef struct {
	Vec3 velocity, location;
	float mass;
} Body;

__device__ float dist2(Vec3 a, Vec3 b) {
	return pow(a.x - b.x, 2) + pow(a.y - b.y, 2) + pow(a.z - b.z, 2);
}

//__device__ Vec3 norm(Vec3 )
__global__
void calculate_forces(int n, Body* bodies)
{
	// everyone should have access to a global body list
	// everyone updates their own body
	// __syncthreads and go to another tick? should i pass in tic levels?

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	float g_const = 10;

	Body* dis_body = &bodies[idx];

	Vec3 force;
	force.x = 0;
	force.y = 0;
	force.z = 0;

	dis_body->mass = 5;

	printf("%f\n", dis_body->mass);
	for(int i = 0; i < n; i++) {

		if(i == idx) {
			continue;
		}

		Body b = bodies[i];

		dis_body->velocity.x += g_const * b.mass * dis_body->mass / dist2(b.location, dis_body->location);
	}

}

int main(void)
{ 
	//int N = 1<<20;
	int N = 10;

	Body* bodies;
	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&bodies, N * sizeof(Body));
	//cudaMallocManaged(&y, N*sizeof(Body));

	// initialize x and y arrays on the host
	/*
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	*/
	for(int i = 0; i < N; i++) {
		bodies[i].velocity.x = 0;
		bodies[i].velocity.y = 0;
		bodies[i].velocity.z = 0;

		bodies[i].location.x = 0;
		bodies[i].location.y = 0;
		bodies[i].location.z = 0;

		bodies[i].mass = 0;
	}

	// Run kernel on 1M elements on the GPU
	//add<<<1, 256>>>(N, x, y);
	calculate_forces<<<1, 256>>>(N, bodies);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	/*
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i]-3.0f));

	std::cout << "Max error: " << maxError << std::endl;
	*/

	for(int i = 0; i < N; i++) {
		std::cout << "x velocity: " << bodies[i].velocity.x << std::endl;
	}
	std::cout << "donezo" << std::endl;

	// Free memory
	hipFree(bodies);
	/*
	cudaFree(x);
	cudaFree(y);
	*/

	return 0;
}
